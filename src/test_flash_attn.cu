#include "hip/hip_runtime.h"
#include "flash_fwd_hdim128_fp16_sm80.cu"
#include <hip/hip_runtime.h>
// #include <torch/torch.h>
#include <iostream>
#include <random>
#include <cmath>
#include <vector>
#include <chrono>



// Modify the comparison function
float compare_results(const cutlass::half_t* gpu_output, const float* cpu_output, 
                     size_t size, float tolerance = 1e-3f) {
    float max_diff = 0.0f;
    int diff_count = 0;
    
    for (size_t i = 0; i < size; i++) {
        float gpu_val = static_cast<float>(gpu_output[i]);
        float diff = std::abs(gpu_val - cpu_output[i]);
        max_diff = std::max(max_diff, diff);
        
        if (diff > 0.01f) {  // Count differences larger than 0.01
            diff_count++;
        }
    }
    
    std::cout << "Number of elements with difference > 0.01: " << diff_count 
              << " out of " << size << " elements (" 
              << (100.0f * diff_count / size) << "%)" << std::endl;
    
    return max_diff;
}
// Helper function to initialize tensor data with sequential numbers
template<typename T>
void initialize_tensor_float(T* data, size_t size) {
    static std::random_device rd;
    static std::mt19937 gen(rd());
    
    // Create uniform distribution between -1 and 1
    std::uniform_real_distribution<float> distribution(-1.0f, 1.0f);
    
    for (size_t i = 0; i < size; i++) {
        // Generate a unique value by combining index and random number
        float random_part = distribution(gen);
        // Combine index and random part to ensure uniqueness
        // Scale down the index to keep values in reasonable range
        float value = random_part + (static_cast<float>(i) / size) * 0.001f;
        // float value = float(i) * 0.1f;
        data[i] = value;
    }

}

// Helper function to zero initialize tensor data
template<typename T>
void zero_initialize_tensor(T* data, size_t size) {
    for (size_t i = 0; i < size; i++) {
        data[i] = static_cast<T>(0.0f);
    }
}

// Helper function to check CUDA errors
// #define CHECK_CUDA(call) do {                                 \
//     hipError_t err = call;                                  \
//     if (err != hipSuccess) {                               \
//         printf("CUDA error at %s %d: %s\n", __FILE__,       \
//                __LINE__, hipGetErrorString(err));          \
//         exit(EXIT_FAILURE);                                 \
//     }                                                       \
// } while(0)

void compute_qk_cpu(
    const cutlass::half_t* q,  // [batch_size, num_heads, seq_len, head_dim]
    const cutlass::half_t* k,  // [batch_size, num_heads, seq_len, head_dim]
    float* output,             // [batch_size, num_heads, seq_len, seq_len]
    uint16_t* indices,         // [batch_size, num_heads, seq_len, topk]
    int batch_size,
    int num_heads,
    int seq_len,
    int head_dim,
    int topk
) {
    // Loop over batches and heads
    for (int b = 0; b < batch_size; b++) {
        for (int h = 0; h < num_heads; h++) {
            // Calculate base offsets for this batch and head
            size_t q_batch_offset = b * (num_heads * seq_len * head_dim);
            size_t k_batch_offset = b * (num_heads * seq_len * head_dim);
            size_t q_head_offset = h * (seq_len * head_dim);
            size_t k_head_offset = h * (seq_len * head_dim);
            size_t o_offset = b * (num_heads * seq_len * seq_len) + h * (seq_len * seq_len);
            size_t idx_offset = b * (num_heads * seq_len * topk) + h * (seq_len * topk);

            // Compute Q * K^T for this batch and head
            for (int i = 0; i < seq_len; i++) {
                // Store dot products and indices for sorting
                std::vector<std::pair<float, uint16_t>> row_values;
                
                for (int j = 0; j < seq_len; j++) {
                    float sum = 0.0f;
                    for (int d = 0; d < head_dim; d++) {
                        size_t q_idx = q_batch_offset + q_head_offset + i * head_dim + d;
                        size_t k_idx = k_batch_offset + k_head_offset + j * head_dim + d;
                        
                        float q_val = static_cast<float>(q[q_idx]);
                        float k_val = static_cast<float>(k[k_idx]);
                        sum += q_val * k_val;
                    }
                    output[o_offset + i * seq_len + j] = sum;
                    row_values.push_back({sum, static_cast<uint16_t>(j)});
                }

                // Sort row values in descending order
                std::sort(row_values.begin(), row_values.end(),
                    [](const auto& a, const auto& b) { return a.first > b.first; });

                // Store top-k indices
                for (int t = 0; t < topk; t++) {
                    indices[idx_offset + i * topk + t] = row_values[t].second;
                }
            }
        }
    }
}



int main() {
    // Define problem dimensions
    const int batch_size = 1;
    const int num_heads = 1;
    const int seq_len = 256;
    const int head_dim = 128;
    const int topk = 16;
    
    // Calculate sizes
    const size_t qk_size = batch_size * num_heads * seq_len * head_dim;
    const size_t o_size = batch_size * num_heads * seq_len * seq_len;
    const size_t topk_size = batch_size * num_heads * seq_len * topk;
    
    // Allocate host memory
    cutlass::half_t *h_q = new cutlass::half_t[qk_size];
    cutlass::half_t *h_k = new cutlass::half_t[qk_size];
    uint16_t *h_ido = new uint16_t[topk_size];
    cutlass::half_t *h_o = new cutlass::half_t[o_size]; 

    
    
    
    // Initialize input tensors
    initialize_tensor_float(h_q, qk_size);
    initialize_tensor_float(h_k, qk_size);
    zero_initialize_tensor(h_o, o_size);
    zero_initialize_tensor(h_ido, topk_size);

    // Allocate device memory
    cutlass::half_t *d_q, *d_k,*d_o;
    uint16_t *d_ido;
    CHECK_CUDA(hipMalloc(&d_q, qk_size * sizeof(cutlass::half_t)));
    CHECK_CUDA(hipMalloc(&d_k, qk_size * sizeof(cutlass::half_t)));
    CHECK_CUDA(hipMalloc(&d_o, o_size * sizeof(cutlass::half_t)));
    CHECK_CUDA(hipMalloc(&d_ido, topk_size * sizeof(uint16_t)));
    
    // Create CUDA stream
    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));
    
    // Copy data to device
    CHECK_CUDA(hipMemcpy(d_q, h_q, qk_size * sizeof(cutlass::half_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_k, h_k, qk_size * sizeof(cutlass::half_t), hipMemcpyHostToDevice));
    // TEST: index o
    // CHECK_CUDA(hipMemcpy(d_o, h_o, o_size * sizeof(cutlass::half_t), hipMemcpyHostToDevice));
    
    // Initialize Flash Attention parameters
    Flash_fwd_params params;
    params.q_ptr = d_q;
    params.k_ptr = d_k;
    params.o_ptr = d_o;
    params.ido_ptr = d_ido;
    // Set dimensions
    params.b = batch_size;
    params.h = num_heads;
    params.seqlen_q = seq_len;
    params.seqlen_k = seq_len;
    params.d = head_dim;
    params.seqlen_q_rounded = seq_len;
    params.seqlen_k_rounded = seq_len;
    params.d_rounded = head_dim;
    params.topk = topk;
    // Set strides
    params.q_batch_stride = num_heads * seq_len * head_dim;
    params.k_batch_stride = num_heads * seq_len * head_dim;
    params.o_batch_stride = num_heads * seq_len * seq_len;
    params.q_head_stride = seq_len * head_dim;
    params.k_head_stride = seq_len * head_dim;
    params.o_head_stride = seq_len * seq_len;
    params.q_row_stride = head_dim;
    params.k_row_stride = head_dim;
    params.o_row_stride = seq_len;
    params.ido_batch_stride = num_heads * seq_len * topk;
    params.ido_head_stride = seq_len * topk;
    params.ido_row_stride = topk;
    
    // Run Flash Attention
    run_mha_fwd_<cutlass::half_t, 128, false>(params, stream);
    
    // Wait for completion
    CHECK_CUDA(hipStreamSynchronize(stream));
    
    // Copy results back to host
    CHECK_CUDA(hipMemcpy(h_o, d_o, o_size * sizeof(cutlass::half_t), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_ido, d_ido, topk_size * sizeof(uint16_t), hipMemcpyDeviceToHost));
    
    std::cout << "\n\nOutput IDO tensor:" << std::endl;
    for (int b = 0; b < batch_size; b++) {
        std::cout << "Batch " << b << ":\n";
        for (int h = 0; h < num_heads; h++) {
            std::cout << "  Head " << h << ":\n";
            for (int s = 0; s < seq_len; s++) {
                std::cout << "    Seq " << s << ": ";
                for (int t = 0; t < topk; t++) {
                    size_t idx = b * params.ido_batch_stride + 
                                h * params.ido_head_stride +
                                s * params.ido_row_stride + t;
                    std::cout << h_ido[idx] << " ";
                }
                std::cout << std::endl;
            }
            std::cout << std::endl;
        }
    }
    std::cout << std::endl;


    // Compute CPU reference implementation
    std::vector<float> cpu_output(o_size);
    std::vector<uint16_t> cpu_indices(topk_size);
    auto cpu_start = std::chrono::high_resolution_clock::now();
    
    compute_qk_cpu(h_q, h_k, cpu_output.data(), cpu_indices.data(),
                  batch_size, num_heads, seq_len, head_dim, topk);
    
    auto cpu_end = std::chrono::high_resolution_clock::now();
    auto cpu_duration = std::chrono::duration_cast<std::chrono::milliseconds>(cpu_end - cpu_start);
    
    // Print timing results
    std::cout << "\nPerformance Results:" << std::endl;
    std::cout << "CPU Execution time: " << cpu_duration.count() << " ms" << std::endl;
    // std::cout << "Speedup: " << static_cast<float>(cpu_duration.count()) / gpu_milliseconds << "x" << std::endl;
    
    // Compare QK results
    float max_diff = compare_results(h_o, cpu_output.data(), o_size);
    
    // Compare indices
    std::cout << "\nComparing indices between CPU and GPU:\n";
    std::cout << "CPU indices:\n";
    for (int b = 0; b < batch_size; b++) {
        std::cout << "Batch " << b << ":\n";
        for (int h = 0; h < num_heads; h++) {
            std::cout << "  Head " << h << ":\n";
            for (int s = 0; s < seq_len; s++) {
                std::cout << "    Seq " << s << ": ";
                for (int t = 0; t < topk; t++) {
                    size_t idx = b * params.ido_batch_stride + 
                                h * params.ido_head_stride +
                                s * params.ido_row_stride + t;
                    std::cout << cpu_indices[idx] << " ";
                }
                std::cout << std::endl;
            }
            std::cout << std::endl;
        }
    }
    
    // Cleanup
    delete[] h_q;
    delete[] h_k;
    // delete[] h_o;
    CHECK_CUDA(hipFree(d_q));
    CHECK_CUDA(hipFree(d_k));
    // CHECK_CUDA(hipFree(d_o));
    CHECK_CUDA(hipFree(d_ido));
    CHECK_CUDA(hipStreamDestroy(stream));
    
    return 0;
} 